#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "types.h"

#define DO_PROG false
#define DO_MAXQ false
#define DO_MAXP false
#define DO_MEMD false

#define PROG_AMT 100

// Depends on input data, priority queue implementation, and adjacency list orders.
#if not DO_MAXQ
#define MAXQ 19430
#endif

#if DO_MAXQ
#define THCT 512u
#else
#define THCT 2560u
#endif
#define TH_PER_BL 64u
#define BLCT (THCT / TH_PER_BL)

#define MAX_STCT 22051
#define MAX_CRSCT 6072
#define MAX_VCT 28123
#define MAX_ECT 118314

#define FP_EPSILON 

#ifdef __INTELLISENSE__
dim3 gridDim;
dim3 blockDim;
uint3 blockIdx;
uint3 threadIdx;
int warpSize;
#endif

typedef float Ftype;

typedef unsigned int* List;

typedef Ftype* FList;

typedef unsigned int* Stack;

// A (min-) binary heap is used, as this program works with sparse graphs.
typedef unsigned int* Queue;

__device__ inline void enqueue(unsigned int& size, Queue q, unsigned int x, FList d)
{
	unsigned int c = size++, P;
	assert(size <= MAX_ECT);
	//Ftype xd = d[x.idx];
	Ftype xd = d[x];
	//while (c > 0 && x.dist < q[p = (c - 1) / 2].dist)
	while (c > 0 && xd < d[q[P = (c - 1)]])
	{
		q[c] = q[P];
		c = P;
	}
	q[c] = x;
}

__device__ inline void dequeue(unsigned int& size, Queue Q, FList D)
{
	unsigned int P = 0, c, l, r, ql, qr, qc;
	Ftype dl, dr;
	assert(size > 0);
	unsigned int x = Q[--size];
	Ftype xd = D[x];
	while ((l = P * 2 + 1) < size && xd > D[Q[c = (r = l + 1) < size && D[Q[l]] > D[Q[r]] ? r : l]])
	//while ((l = p * 2 + 1) < size && xd > (c = (r = l + 1) < size && (dl = D[ql = Q[l]]) > (dr = D[qr = Q[r]]) ? (qc = qr, dr) : (qc = ql, dl)))
	{
		Q[P] = Q[c];
		P = c;
	}
	Q[P] = x;
}

// Implementation using Brandes' algorithm
__global__ void kernelBetcA(
	unsigned int stct,
	unsigned int crsct,
	unsigned int ect,
	List stdeg,
	List crsdeg,
	List* stadj,
	List* crsadj,
	FList weights,
	FList betcaOut,
	Stack* stacks,
	Queue* queues,
	List* prevs,
	List* prevsp,
	List** prev,
	FList* dist,
	FList* sigma,
	FList* delta
#if DO_MAXQ
	, List maxQ
#endif
#if DO_MAXP
	, List maxP
#endif
#if DO_PROG
	, unsigned int* prog
#endif

)
{
	const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int stride = blockDim.x * gridDim.x;

	const unsigned int vct = stct + crsct;

	List* P = prev[index];
	List Ps = prevs[index];
	Stack S = stacks[index];
	Queue Q = queues[index];
	FList D = dist[index];
	FList Sp = sigma[index];
	FList Dep = delta[index];

	{
		List Psp = prevsp[index];
		unsigned int pos = 0;
		for (unsigned int stid = 0; stid < stct; stid++)
		{
			//if (stid == 0)printf("%d %d %d %p %d\n", index, stride, vct, (void*)P, stdeg[index]);
			P[stid] = Psp + pos;
			pos += stdeg[stid];
		}
		for (unsigned int crsid = 0; crsid < crsct; crsid++)
		{
			P[crsid + stct] = Psp + pos;
			pos += crsdeg[crsid];
		}
		assert(pos == 2 * ect);
	}

	unsigned int Ss = 0;
	unsigned int Qs = 0;

	for (unsigned int s = index; s < vct; s += stride)
	{
#if DO_PROG
		unsigned int pr = atomicAdd(prog, PROG_AMT);
		if (pr == 0 || pr / vct > (pr - PROG_AMT) / vct) printf("%3d/%d\n", pr / vct, PROG_AMT);
#endif
#if DO_MAXP
		unsigned int totalP = 0;
#endif

		assert(Ss == 0);
		assert(Qs == 0);
		for (unsigned int t = 0; t < vct; t++)
		{
			Ps[t] = 0;
			Sp[t] = 0.;
			D[t] = -1.;
			Dep[t] = 0.;
		}
		Sp[s] = 1.;
		D[s] = 0.;
		enqueue(Qs, Q, s, D);
#if DO_MAXQ
		if (Qs > maxQ[index]) maxQ[index] = Qs;
#endif
		//printf("Reached %d\n", index);
		while (Qs > 0)
		{
			//unsigned int v = Q->idx;
			//Ftype dv = Q->dist;
			unsigned int v = *Q;
			Ftype dv = D[v];
			dequeue(Qs, Q, D);
			const bool vst = v < stct;
			S[Ss++] = v;
			assert(Ss <= MAX_VCT);
			const unsigned int vdeg = vst ? stdeg[v] : crsdeg[v - stct];
			List vadj = vst ? stadj[v] : crsadj[v - stct];
			Ftype dw;
			if (!vst) dw = dv + weights[v - stct];
			for (unsigned int i = 0; i < vdeg; i++)
			{
				unsigned int w = vadj[i];
				if (vst)
				{
					dw = dv + weights[w];
					w += stct;
				}
				if (D[w] < 0)
				{
					D[w] = dw;
					enqueue(Qs, Q, w, D);
#if DO_MAXQ
					if (Qs > maxQ[index]) maxQ[index] = Qs;
#endif
				}
				assert(D[w] <= dw);
				if (D[w] == dw)
				{
					Sp[w] += Sp[v];
					P[w][Ps[w]++] = v;
#if DO_MAXP
					totalP++;
#endif
				}
			}
		}
#if DO_MAXP
		if (totalP > maxP[index]) maxP[index] = totalP;
#endif
		while (Ss > 0)
		{
			unsigned int w = S[--Ss];
			for (unsigned int i = 0; i < Ps[w]; i++)
			{
				unsigned int v = P[w][i];
				Dep[v] += Sp[v] / Sp[w] * (1 + Dep[w]);
			}
			if (w != s) atomicAdd(betcaOut + w, 0.5 * Dep[w]);
		}
	}
}

__host__ void compBetcA(
	unsigned int stct,
	unsigned int crsct,
	unsigned int ect,
	const unsigned int* stdeg,
	const unsigned int* crsdeg,
	const unsigned int* const* stadj,
	const unsigned int* const* crsadj,
	const Ftype* weights,
	Ftype* betcaOut
)
{
	//hipDeviceProp_t prop; hipGetDeviceProperties(&prop, 0);

	unsigned int vct = stct + crsct;

	hipError_t err;

	List stdeg_d;
	err = hipMalloc(&stdeg_d, stct * sizeof(unsigned int));
	assert(!err);
	err = hipMemcpy(stdeg_d, stdeg, stct * sizeof(unsigned int), hipMemcpyHostToDevice);
	assert(!err);

	List crsdeg_d;
	err = hipMalloc(&crsdeg_d, crsct * sizeof(unsigned int));
	assert(!err);
	err = hipMemcpy(crsdeg_d, crsdeg, crsct * sizeof(unsigned int), hipMemcpyHostToDevice);
	assert(!err);

	List* stadj_d0;
	err = hipMalloc(&stadj_d0, stct * sizeof(List));
	assert(!err);
	List* stadj_d1 = new List[stct];
	for (unsigned int stid = 0; stid < stct; stid++)
	{
		err = hipMalloc(stadj_d1 + stid, stdeg[stid] * sizeof(unsigned int));
		assert(!err);
		err = hipMemcpy(stadj_d1[stid], stadj[stid], stdeg[stid] * sizeof(unsigned int), hipMemcpyHostToDevice);
		assert(!err);
	}
	err = hipMemcpy(stadj_d0, stadj_d1, stct * sizeof(List), hipMemcpyHostToDevice);
	assert(!err);

	List* crsadj_d0;
	err = hipMalloc(&crsadj_d0, crsct * sizeof(List));
	assert(!err);
	List* crsadj_d1 = new List[crsct];
	for (unsigned int crsid = 0; crsid < crsct; crsid++)
	{
		err = hipMalloc(crsadj_d1 + crsid, crsdeg[crsid] * sizeof(unsigned int));
		assert(!err);
		err = hipMemcpy(crsadj_d1[crsid], crsadj[crsid], crsdeg[crsid] * sizeof(unsigned int), hipMemcpyHostToDevice);
		assert(!err);
	}
	err = hipMemcpy(crsadj_d0, crsadj_d1, crsct * sizeof(List), hipMemcpyHostToDevice);
	assert(!err);

	FList weights_d;
	err = hipMalloc(&weights_d, crsct * sizeof(Ftype));
	assert(!err);
	err = hipMemcpy(weights_d, weights, crsct * sizeof(Ftype), hipMemcpyHostToDevice);
	assert(!err);

	FList betca_d;
	err = hipMalloc(&betca_d, vct * sizeof(Ftype));
	assert(!err);
	err = hipMemset(betca_d, 0, vct * sizeof(Ftype));
	assert(!err);

	Stack* stack_d0;
	err = hipMalloc(&stack_d0, THCT * sizeof(Stack));
	assert(!err);
	Stack* stack_d1 = new Stack[THCT];
	for (unsigned int thid = 0; thid < THCT; thid++)
	{
		hipMalloc(stack_d1 + thid, vct * sizeof(unsigned int));
	}
	err = hipMemcpy(stack_d0, stack_d1, THCT * sizeof(Stack), hipMemcpyHostToDevice);
	assert(!err);

	Queue* queue_d0;
	err = hipMalloc(&queue_d0, THCT * sizeof(Queue));
	assert(!err);
	Queue* queue_d1 = new Queue[THCT];
	for (unsigned int thid = 0; thid < THCT; thid++)
	{
#ifdef MAXQ
		err = hipMalloc(queue_d1 + thid, MAXQ * sizeof(unsigned int));
#else
		err = hipMalloc(queue_d1 + thid, (ect + 1) * sizeof(unsigned int));
#endif
		assert(!err);
	}
	err = hipMemcpy(queue_d0, queue_d1, THCT * sizeof(Queue), hipMemcpyHostToDevice);
	assert(!err);

	List* prevs_d0;
	err = hipMalloc(&prevs_d0, THCT * sizeof(List));
	assert(!err);
	List* prevs_d1 = new List[THCT];
	for (unsigned int thid = 0; thid < THCT; thid++)
	{
		err = hipMalloc(prevs_d1 + thid, vct * sizeof(unsigned int));
		assert(!err);
	}
	err = hipMemcpy(prevs_d0, prevs_d1, THCT * sizeof(List), hipMemcpyHostToDevice);
	assert(!err);

	List* prevsp_d0;
	err = hipMalloc(&prevsp_d0, THCT * sizeof(List));
	assert(!err);
	List* prevsp_d1 = new List[THCT];
	for (unsigned int thid = 0; thid < THCT; thid++)
	{
		err = hipMalloc(prevsp_d1 + thid, ect * 2 * sizeof(unsigned int));
		assert(!err);
	}
	err = hipMemcpy(prevsp_d0, prevsp_d1, THCT * sizeof(List), hipMemcpyHostToDevice);
	assert(!err);

	List** prev_d0;
	err = hipMalloc(&prev_d0, THCT * sizeof(List*));
	assert(!err);
	List** prev_d1 = new List * [THCT];
	for (unsigned int thid = 0; thid < THCT; thid++)
	{
		err = hipMalloc(prev_d1 + thid, vct * sizeof(List));
		assert(!err);
	}
	err = hipMemcpy(prev_d0, prev_d1, THCT * sizeof(List*), hipMemcpyHostToDevice);
	assert(!err);

	FList* dist_d0;
	err = hipMalloc(&dist_d0, THCT * sizeof(FList));
	assert(!err);
	FList* dist_d1 = new FList[THCT];
	for (unsigned int thid = 0; thid < THCT; thid++)
	{
		err = hipMalloc(dist_d1 + thid, vct * sizeof(Ftype));
		assert(!err);
	}
	err = hipMemcpy(dist_d0, dist_d1, THCT * sizeof(FList), hipMemcpyHostToDevice);
	assert(!err);

	FList* sigma_d0;
	err = hipMalloc(&sigma_d0, THCT * sizeof(FList));
	assert(!err);
	FList* sigma_d1 = new FList[THCT];
	for (unsigned int thid = 0; thid < THCT; thid++)
	{
		err = hipMalloc(sigma_d1 + thid, vct * sizeof(Ftype));
		assert(!err);
	}
	err = hipMemcpy(sigma_d0, sigma_d1, THCT * sizeof(FList), hipMemcpyHostToDevice);
	assert(!err);

	FList* delta_d0;
	err = hipMalloc(&delta_d0, THCT * sizeof(FList));
	assert(!err);
	FList* delta_d1 = new FList[THCT];
	for (unsigned int thid = 0; thid < THCT; thid++)
	{
		err = hipMalloc(delta_d1 + thid, vct * sizeof(Ftype));
		assert(!err);
	}
	err = hipMemcpy(delta_d0, delta_d1, THCT * sizeof(FList), hipMemcpyHostToDevice);
	assert(!err);

#if DO_PROG
	unsigned int* prog_d;
	err = hipMalloc(&prog_d, sizeof(unsigned int));
	assert(!err);
	err = hipMemset(prog_d, 0, sizeof(unsigned int));
	assert(!err);
#endif

#if DO_MAXQ
	List maxQ_d;
	err = hipMalloc(&maxQ_d, THCT * sizeof(unsigned int));
	assert(!err);
	err = hipMemset(maxQ_d, 0, THCT * sizeof(unsigned int));
	assert(!err);
#endif

#if DO_MAXP
	List maxP_d;
	err = hipMalloc(&maxP_d, THCT * sizeof(unsigned int));
	assert(!err);
	err = hipMemset(maxP_d, 0, THCT * sizeof(unsigned int));
	assert(!err);
#endif

	err = hipDeviceSynchronize();
	assert(!err);

#if DO_MEMD
	size_t free, total;
	err = hipMemGetInfo(&free, &total);
	assert(!err);
	printf("%llu free\n%llu total\n", free, total);
#endif

	kernelBetcA << <BLCT, TH_PER_BL >> > (
		stct,
		crsct,
		ect,
		stdeg_d,
		crsdeg_d,
		stadj_d0,
		crsadj_d0,
		weights_d,
		betca_d,
		stack_d0,
		queue_d0,
		prevs_d0,
		prevsp_d0,
		prev_d0,
		dist_d0,
		sigma_d0,
		delta_d0
#if DO_MAXQ
		, maxQ_d
#endif
#if DO_MAXP
		, maxP_d
#endif
#if DO_PROG
		, prog_d
#endif
		);
	err = hipPeekAtLastError();
	assert(!err);
	err = hipDeviceSynchronize();
	assert(!err);

	err = hipFree(stdeg_d);
	assert(!err);

	err = hipFree(crsdeg_d);
	assert(!err);

	err = hipFree(stadj_d0);
	assert(!err);
	for (unsigned int stid = 0; stid < stct; stid++)
	{
		err = hipFree(stadj_d1[stid]);
		assert(!err);
	}
	delete[] stadj_d1;

	err = hipFree(crsadj_d0);
	assert(!err);
	for (unsigned int crsid = 0; crsid < crsct; crsid++)
	{
		err = hipFree(crsadj_d1[crsid]);
		assert(!err);
	}
	delete[] crsadj_d1;

	err = hipFree(weights_d);
	assert(!err);

	err = hipMemcpy(betcaOut, betca_d, vct * sizeof(Ftype), hipMemcpyDeviceToHost);
	assert(!err);
	err = hipFree(betca_d);
	assert(!err);

	err = hipFree(stack_d0);
	assert(!err);
	for (unsigned int thid = 0; thid < THCT; thid++)
	{
		err = hipFree(stack_d1[thid]);
		assert(!err);
	}
	delete[] stack_d1;

	err = hipFree(queue_d0);
	assert(!err);
	for (unsigned int thid = 0; thid < THCT; thid++)
	{
		err = hipFree(queue_d1[thid]);
		assert(!err);
	}
	delete[] queue_d1;

	err = hipFree(prevs_d0);
	assert(!err);
	for (unsigned int thid = 0; thid < THCT; thid++)
	{
		err = hipFree(prevs_d1[thid]);
		assert(!err);
	}
	delete[] prevs_d1;

	err = hipFree(prevsp_d0);
	assert(!err);
	for (unsigned int thid = 0; thid < THCT; thid++)
	{
		err = hipFree(prevsp_d1[thid]);
		assert(!err);
	}
	delete[] prevsp_d1;

	err = hipFree(prev_d0);
	assert(!err);
	for (unsigned int thid = 0; thid < THCT; thid++)
	{
		err = hipFree(prev_d1[thid]);
		assert(!err);
	}
	delete[] prev_d1;

	err = hipFree(dist_d0);
	assert(!err);
	for (unsigned int thid = 0; thid < THCT; thid++)
	{
		err = hipFree(dist_d1[thid]);
		assert(!err);
	}
	delete[] dist_d1;

	err = hipFree(sigma_d0);
	assert(!err);
	for (unsigned int thid = 0; thid < THCT; thid++)
	{
		err = hipFree(sigma_d1[thid]);
		assert(!err);
	}
	delete[] sigma_d1;

	err = hipFree(delta_d0);
	assert(!err);
	for (unsigned int thid = 0; thid < THCT; thid++)
	{
		err = hipFree(delta_d1[thid]);
		assert(!err);
	}
	delete[] delta_d1;

#if DO_PROG
	err = hipFree(prog_d);
	assert(!err);
#endif

#if DO_MAXQ
	List maxQ = new unsigned int[THCT];
	err = hipMemcpy(maxQ, maxQ_d, THCT * sizeof(unsigned int), hipMemcpyDeviceToHost);
	assert(!err);
	err = hipFree(maxQ_d);
	assert(!err);
	unsigned int maxQPt = 0;
	for (unsigned int i = 0; i < THCT; i++)
		if (maxQ[i] > maxQPt)
			maxQPt = maxQ[i];
	delete[] maxQ;
	printf("Max Queue Size: %d\n", maxQPt);
#endif

#if DO_MAXP
	List maxP = new unsigned int[THCT];
	err = hipMemcpy(maxP, maxP_d, THCT * sizeof(unsigned int), hipMemcpyDeviceToHost);
	assert(!err);
	err = hipFree(maxP_d);
	assert(!err);
	unsigned int maxPPt = 0;
	for (unsigned int thid = 0; thid < THCT; thid++)
		if (maxP[thid] > maxPPt) maxPPt = maxP[thid];
	delete[] maxP;
	printf("Max Prev Size: %d\n", maxPPt);
#endif

}